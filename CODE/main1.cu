



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include <cutil.h>
#include<time.h>
#include <iostream>
#include <string>
#include <vector>



#define DEBUG_MODE 0
#define MASTER_PROCESSOR_RANK 0
#define DELTA_T 10
#define INPUT_FILE_DATA "inputfile.txt"

#define ToalTimeStep 10000
const float DEG2RAD = 3.14159/180;

using namespace std;
double  Gconstant = 6.6674 * pow(10,-11);
// #define Gconstant 6.6674 * pow(10,-11)


typedef long long int lld;

// lld n;
lld numeroflines ;


// double *arrx1 *arry2;


// __global__ void kernelcomputenewvel(double* idx ,double *masslist, double *velocitylistx,double *velocitylisty, double *newvelocitylistx,double *newvelocitylisty,double *postionx,double *postiony,double *newpostionx,double *newpostiony ,lld threadcunt )
__global__ void kernelcomputenewvel(double masslist[], double velocitylistx[],double velocitylisty[], double postionx[],double postiony[],double Forcex1[],double Forcey2[] ,lld idx ,lld threadcunt,lld n )
{
    // lld threadid = blockDim.x * blockIdx.x + threadIdx.x;
    lld threadid =  threadIdx.x;

    double m2 ,r1vector,r2vector,rdiffector;
    // double m1 = masslist[idx];
    // double curentpos_1x = postionx[idx];
    // double curentpos_1y = postiony[idx];

    // double curentvel_1x = velocitylistx[idx];
    // double curentvel_1y = velocitylisty[idx];

    Forcex1[threadid] = 0;
    Forcey2[threadid] = 0 ;


    for(lld i1 = threadid;i1<n;i1+=threadcunt)
    {
        // c1temp[i1] = a1[i1]+b1[i1];
        if(i1!=idx)
        {
            m2 = masslist[i1] ;
            r1vector = postionx[i1] - postionx[idx];
            r2vector = postiony[i1] - postiony[idx];

            rdiffector = r1vector*r1vector+r2vector*r2vector;
            rdiffector = pow(rdiffector,1.5);

            // Forcey2[threadid] = Forcey2[threadid] + (((Gconstant*masslist[idx]*m2)/rdiffector)*r2vector) ;
            // Forcex1[threadid] = Forcex1[threadid] + (((Gconstant*masslist[idx]*m2)/rdiffector)*r1vector) ;
             
            // Forcey2[threadid] = Forcey2[threadid] + (((masslist[idx]*m2)/rdiffector)*r2vector) ;
            // Forcex1[threadid] = Forcex1[threadid] + (((masslist[idx]*m2)/rdiffector)*r1vector) ;


            Forcey2[threadid] = Forcey2[threadid] + ((m2/rdiffector)*r2vector) ;
            Forcex1[threadid] = Forcex1[threadid] + ((m2/rdiffector)*r1vector) ;

        }
    }

    // __syncthreads(); // barrier point 


    //update all 
    // if(threadid ==0 )
    // {

    // }

}









int main(int argc, char **argv)
{
    int x1,x2;
    int deltavartobeused;
    // cout<<argc;
    double deltat = DELTA_T;

    if (argc-1==2)
    {
        //argv[1] number of threads ,number of bodies
         x1 = atoi(argv[1]);
         x2 = atoi(argv[2]);
        // cout<<x1;
        // if(argc-1==3)
        // {
        //     deltat = atoi
        // }

    }




    FILE * fp = fopen(INPUT_FILE_DATA, "r");
    if (!fp) {
      printf("Error opening input file.\n");
      exit(1);
    }

    FILE *fp2,*fp3,*fp4;

    // fp4 = fopen("justcoordinates.txt","w+");
    fp2 = fopen("outputtxt.txt","w");
    fp3 = fopen("outputtonlytimedone.txt","w");
    long long int nobj;
     double m1,newposxi1,newposyi1,m2,r1vector,r2vector,rdiffector;
    double Forcex1,Forcey2;
    
    double accelerationx,accerlerationy;
    double tempvx1,tempvy2;
    double average_kernel_time = 0;



    //host particle storage 
    double posx[120000];
    double posy[120000];
    double mass[120000];
    double velex[120000];
    double veley[120000];
    double *forecearrayx,*forecearrayy;
    double *hoforecearrayx,*hoforecearrayy;



    //device particle storage
    double *dposx;
    double *dposy;
    double *dmass;
    double *dvelex;
    double *dveley;

    lld maxnumberofobjs ;
    //temporary store
    // double *dtempposx;
    // double *dtempposy;
    // double *dtempvelex;
    // double *dtempveley;
    
    // lld threads_in_block = 1024;
    lld threads_in_block = x1;




    // hoforecearrayx = new double[]  ;

    long long int array_siez,numer_of_lines;
    
    numer_of_lines = ToalTimeStep/deltat;

      fscanf(fp, "%lld", &maxnumberofobjs);
      cout<<"MAXIMUM NUMBER of objects "<<maxnumberofobjs<<endl;
    //   nobj = 2048;
    //   nobj = 5040;
    nobj = x2 ; 

    //   n = nobj ;
      cout<<"Selected number of objects "<<nobj<<endl;
      cout<<"Number of Threads "<<threads_in_block<<endl;
      cout<<"Deltat "<<deltat<<endl;


        fp4 = fopen("justcoordinates.txt","w+");
        
        fprintf(fp4,"%lld",nobj);

        fprintf(fp4,"\n");
        fprintf(fp4,"%lld",numer_of_lines);
        fprintf(fp4,"\n");

        // fclose(fp4);
    for (lld rep = 0; rep < nobj; rep++)
    {
        // cin>>posx[rep]>>posy[rep]>>mass[rep];

        fscanf(fp, "%lf %lf %lf", &posx[rep],&posy[rep],&mass[rep]);
        velex[rep] = veley[rep] = 0;
    }

    if(DEBUG_MODE>=2)
    {
        for (lld rep = 0; rep < nobj; rep++)
        {
            // cin>>posx[rep]>>posy[rep]>>mass[rep];
    
            fprintf(fp3, "%lf %lf %lf\n", posx[rep],posy[rep],mass[rep]);
            // velex[rep] = veley[rep] = 0;
        }

    }
    


    float et;
    hipEvent_t start, stop;
  
    clock_t startc, end;
    startc = clock();



    // fclose(fp);
    array_siez = nobj;
     lld array_bytes = array_siez*sizeof(double);
     lld threadsize = threads_in_block*sizeof(double);
    hoforecearrayx = (double*)malloc(threadsize);
    hoforecearrayy = (double*)malloc(threadsize);
    // hoforecearrayx = new double[threads_in_block];
    // hoforecearrayy = new double[threads_in_block];



    hipMalloc((void **)&dposx,array_bytes );
    hipMalloc((void **)&dposy,array_bytes );
    hipMalloc((void **)&dmass,array_bytes );
    hipMalloc((void **)&dvelex,array_bytes );
    hipMalloc((void **)&dveley,array_bytes );
    hipMalloc((void **)&forecearrayx,threadsize );
    hipMalloc((void **)&forecearrayy,threadsize );










    for (lld timestep = 0; timestep*deltat < ToalTimeStep ; timestep++)
        {
            // fprintf(fp2,"%lf",timestep*deltat);
            // fprintf(fp2,"\n");
       
            for(lld i1 = 0 ;i1<nobj; i1++)
            {
                //each body send data to kernel 
                //a parallel compute vel

                //cpoy data to gpu

                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start,0);
                hipEventRecord(stop,0);
            

               

            hipMemcpy(dposx,posx,array_bytes,hipMemcpyHostToDevice);
            hipMemcpy(dposy,posy,array_bytes,hipMemcpyHostToDevice);
            hipMemcpy(dmass,mass,array_bytes,hipMemcpyHostToDevice);
            hipMemcpy(dvelex,velex,array_bytes,hipMemcpyHostToDevice);
            hipMemcpy(dveley,veley,array_bytes,hipMemcpyHostToDevice);
            // cudaMemcpy(dveley,veley,array_bytes,cudaMemcpyHostToDevice);
            // cudaMemcpy(dveley,veley,array_bytes,cudaMemcpyHostToDevice);




        kernelcomputenewvel<<<1,threads_in_block>>>(dmass,dvelex,dveley,dposx,dposy,forecearrayx,forecearrayy,i1,threads_in_block,nobj);
// __global__ void kernelcomputenewvel(double idx ,double masslist[], double velocitylistx[],double velocitylisty[], double postionx[],double postiony[],lld threadcunt,double Forcex1[],double Forcey2[] )
            

            // cudaMemcpy(hoforecearrayx,forecearrayx,array_bytes,cudaMemcpyDeviceToHost);
            // cudaMemcpy(hoforecearrayy,forecearrayy,array_bytes,cudaMemcpyDeviceToHost);


            hipMemcpy(hoforecearrayx,forecearrayx,threadsize,hipMemcpyDeviceToHost);
            hipMemcpy(hoforecearrayy,forecearrayy,threadsize,hipMemcpyDeviceToHost);

            hipEventSynchronize(stop);
        hipEventElapsedTime(&et, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);


            if(DEBUG_MODE>=1)
            {

                printf("\nGPU Time to generate kernel %lld:  %f  \n", timestep*nobj+i1,et);
            }
            average_kernel_time+=et; 

            Forcex1 = 0;Forcey2 = 0;
            for(int ij1=0;ij1<threads_in_block;ij1++)
            {
                Forcex1 += hoforecearrayx[ij1];
                Forcey2 += hoforecearrayy[ij1] ;
            }
           

            //once done update with new vector

            m1 = mass[i1];

            // accelerationx = Gconstant*Forcex1/m1;
            accelerationx = Gconstant*Forcex1;

            tempvx1 = velex[i1] + accelerationx*deltat;
            posx[i1] += (velex[i1]*deltat+(0.5*accelerationx*deltat*deltat) );
            velex[i1] = tempvx1;

    //moement in y direction 
            
            // accerlerationy = Gconstant*Forcey2/m1;
            accerlerationy = Gconstant*Forcey2;

            tempvy2 = veley[i1] + accerlerationy*deltat;
            posy[i1] += (veley[i1]*deltat+(0.5*accerlerationy*deltat*deltat) );
            veley[i1] = tempvy2;

             if(DEBUG_MODE>=2)
                {
                  cout<<" acceleration at each step of objects"<<accelerationx<<" "<< accerlerationy << endl;
                }




            }

            
            //write postion x ,position y


            for (lld xdata = 0; xdata < nobj; xdata++)
            {
    
                // printf("%lf,",posx[xdata]);
                // fprintf(fp2,"%lf ",posx[xdata]);
                fprintf(fp4,"%lf ",posx[xdata]);
    
            }
            fprintf(fp4,"\n");



            for (lld ydata = 0; ydata < nobj; ydata++)
            {
                // printf("%lf,",posx[ydata]);
                
                fprintf(fp4,"%lf ",posy[ydata]);
    
            }
            fprintf(fp4,"\n");




        }



        
    end = clock();

    printf("\naverage kernel time :  %f  \n", float((deltat*average_kernel_time) /(ToalTimeStep*nobj) ) );
    printf("\nTotal time:  %f  \n", float(end-startc) );











        hipFree(dposx );
        hipFree(dposy );
        hipFree(dmass );
        hipFree(dvelex );
        hipFree(dveley );
        hipFree(forecearrayx );
        hipFree(forecearrayy );
    

        // cudaFree(d_in1);
        // cudaFree(d_in2);
        // cudaFree(d_out);
        free(hoforecearrayx);
        free(hoforecearrayy);
        // hoforecearrayx = new double[threads_in_block];
    // hoforecearrayy = new double[threads_in_block];

    fclose(fp);
    fclose(fp2);
    fclose(fp3);
    fclose(fp4);

        return 0;



}






